#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // 假设这里有一些计算
    for (int i = 0; i < 1000; ++i) {
        idx *= 2;
    }
}

int main() {
    // 定义并创建CUDA事件
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // 预热运行
    kernel << <256, 256 >> > ();
    hipDeviceSynchronize();

    // 记录起始事件
    hipEventRecord(start, 0);

    // 调用CUDA核函数
    kernel << <256, 256 >> > ();

    // 记录结束事件
    hipEventRecord(stop, 0);

    // 等待事件完成
    hipEventSynchronize(stop);

    // 计算时间差
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    // 打印执行时间
    std::cout << "Elapsed time: " << elapsedTime << " ms" << std::endl;

    // 销毁CUDA事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
